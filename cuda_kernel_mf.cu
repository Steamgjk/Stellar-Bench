#include "hip/hip_runtime.h"
#include "cuda_kernel_mf.h"
__global__ void cuda_mf(double *row_eles, double* col_eles, int dim, int row_height, int col_height, double* rating_entries, int* random_row_id, int* random_col_id, int wn){
	int tid =  threadIdx.x;
	int r_rx = random_row_id[tid];
	int r_cx = random_col_id[tid];
	int i, j, k;
	double yita = 0.2;
	double theta = 0.2;
	double err = 0;
	for( i = r_rx; i < row_height; i+= wn){
		for(j = r_cx; j < col_height; j+= wn ){
			err = rating_entries[i*col_height+j];
			if(err == 0){
				continue;
			}
			for(k = 0; k < dim; k++){
				err -= row_eles[i*dim+k] * col_eles[j*dim+k];
			}

	        for (k = 0; k < dim; ++k)
	        {
	        	row_eles[i*dim+k] += yita * (err * col_eles[j*dim+k] - theta * row_eles[i*dim+k]);

	            col_eles[j*dim+k] += yita * (err * row_eles[i*dim+k] - theta* col_eles[j*dim+k]);
	        }
		}
	}
}