#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "gpu_mf.h"
using namespace std;
void gpu_mf(double *row_eles, double* col_eles, int dim, int row_height, int col_height, double* rating_entries, int* random_row_id, int* random_col_id, int wn){
	double* d_row_eles, *d_col_eles, *d_rating_entries;
	int* d_random_row_id, *d_random_col_id;
	hipMalloc((void**)&d_row_eles, sizeof(double)* row_height*dim);
	hipMalloc((void**)&d_col_eles, sizeof(double)* col_height*dim);
	hipMalloc((void**)&d_rating_entries, sizeof(double)* row_height*col_height);
	hipMalloc((void**)&d_random_row_id, sizeof(int)* wn);
	hipMalloc((void**)&d_random_col_id, sizeof(int)* wn);

	hipMemcpy(d_row_eles, row_eles, sizeof(double)* row_height*dim,  hipMemcpyHostToDevice);
	hipMemcpy(d_col_eles, col_eles, sizeof(double)* col_height*dim,  hipMemcpyHostToDevice);
	hipMemcpy(d_rating_entries, rating_entries, sizeof(double)* row_height*col_height,  hipMemcpyHostToDevice);

	int iter = 0;
	int iter_num = 100;
	int i = 0;
	for(iter = 0;  iter < iter_num; iter++){
		for( i = 0; i < wn; i++){
			random_row_id[i] = i;
			random_col_id[i] =i;
		}
		random_shuffle(random_row_id, random_row_id+wn);
		random_shuffle(random_col_id, random_col_id+wn);
		hipMemcpy(d_random_row_id, random_row_id, sizeof(int)* wn, hipMemcpyHostToDevice);
		hipMemcpy(d_random_col_id, random_col_id, sizeof(int)* wn, hipMemcpyHostToDevice);
		cuda_mf<<<1, wn>>>(d_row_eles, d_col_eles, dim,row_height,col_height, d_rating_entries, d_random_row_id, d_random_col_id, wn);
	}
	hipMemcpy(row_eles, d_row_eles, sizeof(double)* row_height*dim, hipMemcpyDeviceToHost);
	hipMemcpy(col_eles, d_col_eles, sizeof(double)* col_height*dim, hipMemcpyDeviceToHost);



}